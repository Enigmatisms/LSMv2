#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include "cuda_err_check.hpp"
#include "cast_kernel.hpp"

#define PREPROCESS_BLOCK 4

float* point_angles = nullptr, *sorted_angles = nullptr;
bool* next_valid = nullptr;
int all_point_num = 0;              // set in memAllocator

extern  "C" {
void deallocatePoints() {
    CUDA_CHECK_RETURN(hipFree(point_angles));
    CUDA_CHECK_RETURN(hipFree(sorted_angles));
    CUDA_CHECK_RETURN(hipFree(next_valid));
}

void updatePointInfo(const Vec2* const meshes, const char* const nexts, int point_num, bool initialized) {
    copy2ConstMem(meshes, nexts, point_num);
    if (initialized == true)
        deallocatePoints();
    CUDA_CHECK_RETURN(hipMalloc((void**) &point_angles, sizeof(float) * point_num));
    CUDA_CHECK_RETURN(hipMalloc((void**) &sorted_angles, sizeof(float) * point_num));
    CUDA_CHECK_RETURN(hipMalloc((void**) &next_valid, sizeof(bool) * point_num));
    all_point_num = point_num;
}

void shadowCasting(const Vec3& pose, Vec2* const host_output, int& point_num) {
    const int thread_per_block = static_cast<int>(std::ceil(static_cast<float>(all_point_num) / PREPROCESS_BLOCK));
    backCullPreprocessKernel<<< PREPROCESS_BLOCK, thread_per_block >>> (pose.x, pose.y, all_point_num, point_angles, next_valid);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // sorting rays
    CUDA_CHECK_RETURN(hipMemcpy(sorted_angles, point_angles, all_point_num * sizeof(float), hipMemcpyDeviceToDevice));
    thrust::sort(thrust::device, sorted_angles, sorted_angles + all_point_num, thrust::less<float>());
    const int invalid_bound = thrust::lower_bound(thrust::device, sorted_angles, sorted_angles + all_point_num, 1e2, thrust::less<float>()) - sorted_angles;
    /// duplicate valid rays
    float* actual_rays = nullptr, *output_depth = nullptr;
    Vec2* out_pts = nullptr;
    const int actual_ray_num = invalid_bound << 1;              // double the number of valid rays (left & right extensions)
    const size_t ray_size = sizeof(float) * actual_ray_num;
    CUDA_CHECK_RETURN(hipMalloc((void **) &actual_rays, sizeof(float) * actual_ray_num));
    CUDA_CHECK_RETURN(hipMalloc((void **) &out_pts, sizeof(Vec2) * actual_ray_num));
    size_t ray_block_num = static_cast<size_t>(std::ceil(static_cast<float>(actual_ray_num) / 16));
    size_t seg_block_num = static_cast<size_t>(std::ceil(static_cast<float>(all_point_num) / 64));
    simpleDuplicateKernel<<< 1, invalid_bound >>> (sorted_angles, actual_rays);
    CUDA_CHECK_RETURN(hipMalloc((void **) &output_depth, ray_size * seg_block_num));
    
    /// get ray - mesh segment intersections. Notice that point_num (all_point_num) equals number of segment
    const size_t shared_memory_size = (sizeof(int) << 4);
    hipStream_t streams[4];
    for (short i = 0; i < 4; i++)
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    for (size_t i = 0, stream_idx = 0; i < seg_block_num; i += 4) {				// 面片
        for (size_t j = 0; j < ray_block_num; j += 4, stream_idx++) {			// 光线
            dim3 dimGrid(4, 4);
            dim3 dimBlock(16, 64);
            pointIntersectKernel<<<dimGrid, dimBlock, shared_memory_size, streams[stream_idx % 4]>>>(
                actual_rays, point_angles, next_valid, output_depth, pose.x, pose.y, all_point_num, actual_ray_num, j, i
            );
        }
    }
    /// output, cleaning up
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    depth2PointKernel<<<1, actual_ray_num>>>(output_depth, actual_rays, seg_block_num, pose.x, pose.y, out_pts);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    CUDA_CHECK_RETURN(hipMemcpy(host_output, out_pts, sizeof(Vec2) * actual_ray_num, hipMemcpyDeviceToHost));
    for (int i = 0; i < 4; i++)
        hipStreamDestroy(streams[i]);
    CUDA_CHECK_RETURN(hipFree(out_pts));
    CUDA_CHECK_RETURN(hipFree(actual_rays));
    CUDA_CHECK_RETURN(hipFree(output_depth));
    point_num = actual_ray_num;
}
}