#include "hip/hip_runtime.h"
#include <numeric>
#include "host_func.hpp"

short *sid_ptr = nullptr, *eid_ptr = nullptr;
float *all_segments = nullptr, *angles_ptr = nullptr, *dists_ptr = nullptr, *final_ranges, *oct_ranges;
bool *flag_ptr = nullptr;
size_t total_seg_num = 0;

__host__ __forceinline__ void intializeFixed(int num_ray) {
    CUDA_CHECK_RETURN(hipHostMalloc((void **) &all_segments, 8192 * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &final_ranges, num_ray * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &oct_ranges, (num_ray << 3) * sizeof(float)));
}

__host__ __forceinline__ void deallocateFixed() {
    CUDA_CHECK_RETURN(hipHostFree(all_segments));
    CUDA_CHECK_RETURN(hipFree(final_ranges));
    CUDA_CHECK_RETURN(hipFree(oct_ranges));
}

__host__ void unwrapMeshes(const Meshes& meshes) {
    size_t mesh_point_cnt = 0;
    total_seg_num = 0;
    for (const Mesh& m: meshes) {
        size_t max_size = m.size() - 1;
        all_segments[mesh_point_cnt++] = m.front().x();
        all_segments[mesh_point_cnt++] = m.front().y();
        for (size_t i = 1; i < max_size; i++) {
            const Eigen::Vector2d& p = m[i];
            float x = p.x(), y = p.y();
            all_segments[mesh_point_cnt++] = x;
            all_segments[mesh_point_cnt++] = y;
            all_segments[mesh_point_cnt++] = x;
            all_segments[mesh_point_cnt++] = y;
        }
        all_segments[mesh_point_cnt++] = m.back().x();
        all_segments[mesh_point_cnt++] = m.back().y();
        total_seg_num += (m.size() - 1);
    }
    updateSegments(all_segments, mesh_point_cnt << 2);
    CUDA_CHECK_RETURN(hipFree(sid_ptr));
    CUDA_CHECK_RETURN(hipFree(eid_ptr));
    CUDA_CHECK_RETURN(hipFree(angles_ptr));
    CUDA_CHECK_RETURN(hipFree(dists_ptr));
    CUDA_CHECK_RETURN(hipFree(flag_ptr));

    CUDA_CHECK_RETURN(hipMalloc((void **) &sid_ptr, total_seg_num * sizeof(short)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &eid_ptr, total_seg_num * sizeof(short)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &angles_ptr, total_seg_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &dists_ptr, total_seg_num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &flag_ptr, total_seg_num * sizeof(bool)));
}

__host__ void rayTraceRenderCpp(const Eigen::Vector3d& lidar_param, const Eigen::Vector3d& pose, std::vector<float>& range) {
    // 对于静态地图而言，由于场景无需频繁update，unwrapMeshes函数调用频率低，则可以省略内存allocation操作
    const Vec3 lidar_vec(lidar_param.x(), lidar_param.y(), lidar_param.z()), pose_vec(pose.x(), pose.y(), pose.z());
    const short ray_num = lidar_vec.get_ray_num(), lidar_ray_blocks = ray_num / DEPTH_DIV_NUM;
    const short num_blocks = (total_seg_num >> 7);          // 面片数 / 128
    preProcess<<<num_blocks, 128>>>(sid_ptr, eid_ptr, angles_ptr, dists_ptr, flag_ptr, ray_num, total_seg_num, lidar_vec, pose_vec);
    hipStream_t streams[8];
    for (short i = 0; i < 8; i++)
        hipStreamCreateWithFlags(&streams[i],hipStreamNonBlocking);
    const short segment_per_block = static_cast<short>(ceil(0.125f * total_seg_num)),
                last_block_seg_num = short(total_seg_num) - 7 * segment_per_block;
    for (short i = 0; i < 8; i++) {
        // 需要计算shared memory大小
        // local segements的大小是 (4B * 4 * len(angles)) / 8 + DEPTH_DIV_NUM * 4B (深度图分区) + (1B * len(flags) / 8) + padding

        // 最后由于bool是单字节的类型，需要padding到4的整数倍字节数
        size_t shared_mem_size = (segment_per_block << 4) + (DEPTH_DIV_NUM << 2) + segment_per_block + 4 - segment_per_block % 4;
        rayTraceKernel<<<lidar_ray_blocks, DEPTH_DIV_NUM, shared_mem_size, streams[i]>>>(
            sid_ptr, eid_ptr, angles_ptr, dists_ptr, flag_ptr, i, segment_per_block, 
            ((i < 7) ? segment_per_block : last_block_seg_num), &oct_ranges[i * ray_num], lidar_vec, pose_vec
        );
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++)
        hipStreamDestroy(streams[i]);
    getMininumRangeKernel<<<lidar_ray_blocks, DEPTH_DIV_NUM>>>(oct_ranges, final_ranges, ray_num);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
}
